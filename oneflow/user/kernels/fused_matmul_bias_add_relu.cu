#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/include/primitive/matmul.h"
#include "oneflow/core/common/optional.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include <hip/hip_runtime.h>

namespace oneflow {

namespace {

ep::primitive::BlasTransposeType GetBlasTransposeType(bool transpose) {
  return transpose ? ep::primitive::BlasTransposeType::T : ep::primitive::BlasTransposeType::N;
}

template<typename Context>
ep::primitive::BlasTransposeType GetBlasTransposeType(Context* ctx, const std::string& attr) {
  return GetBlasTransposeType(ctx->template Attr<bool>(attr));
}

Optional<hipDataType> OptCudaDataType(DataType data_type) {
  switch (data_type) {
    case kFloat: return HIP_R_32F;
    case kDouble: return HIP_R_64F;
    case kFloat16: return HIP_R_16F;
#if CUDA_VERSION >= 11000
    case kBFloat16: return HIP_R_16BF;
#endif  // CUDA_VERSION >= 11000
    default: return NullOpt;
  }
}

hipDataType GetCudaDataType(DataType data_type) {
  auto cuda_data_type = OptCudaDataType(data_type);
  CHECK(cuda_data_type.has_value());
  return cuda_data_type.value_or(HIP_R_32F);
}

hipblasComputeType_t GetComputeType(DataType data_type) {
  switch (data_type) {
    case kFloat: return HIPBLAS_COMPUTE_32F;
    case kDouble: return HIPBLAS_COMPUTE_64F;
    case kFloat16: return HIPBLAS_COMPUTE_32F;
#if CUDA_VERSION >= 11000
    case kBFloat16: return HIPBLAS_COMPUTE_32F;
#endif  // CUDA_VERSION >= 11000
    default: UNIMPLEMENTED(); return HIPBLAS_COMPUTE_32F;
  }
}

union CublasScalarParameter {
  double d;
  float s;
};

CublasScalarParameter GetCublasScalarParameter(Scalar scalar, hipblasComputeType_t compute_type) {
  CublasScalarParameter sp{};
  if (compute_type == HIPBLAS_COMPUTE_64F) {
    sp.d = scalar.Value<double>();
  } else if (compute_type == HIPBLAS_COMPUTE_32F) {
    sp.s = scalar.Value<float>();
  } else {
    UNIMPLEMENTED();
  }
  return sp;
}

void InferMatmulMNK(const ShapeView& a_shape, const ShapeView& b_shape, const ShapeView& c_shape,
                    ep::primitive::BlasTransposeType transpose_a,
                    ep::primitive::BlasTransposeType transpose_b, size_t* m, size_t* n, size_t* k) {
  const int64_t num_a_axes = a_shape.NumAxes();
  CHECK_GE(num_a_axes, 2);
  const int64_t num_b_axes = b_shape.NumAxes();
  CHECK_GE(num_b_axes, 2);
  const int64_t num_c_axes = c_shape.NumAxes();
  CHECK_GE(num_c_axes, 2);
  if (transpose_a == ep::primitive::BlasTransposeType::N) {
    *m = a_shape.At(num_a_axes - 2);
    *k = a_shape.At(num_a_axes - 1);
  } else if (transpose_a == ep::primitive::BlasTransposeType::T) {
    *m = a_shape.At(num_a_axes - 1);
    *k = a_shape.At(num_a_axes - 2);
  } else {
    UNIMPLEMENTED();
  }
  if (transpose_b == ep::primitive::BlasTransposeType::N) {
    CHECK_EQ(b_shape.At(num_b_axes - 2), *k);
    *n = b_shape.At(num_b_axes - 1);
  } else if (transpose_b == ep::primitive::BlasTransposeType::T) {
    CHECK_EQ(b_shape.At(num_b_axes - 1), *k);
    *n = b_shape.At(num_b_axes - 2);
  } else {
    UNIMPLEMENTED();
  }
  CHECK_EQ(c_shape.At(num_c_axes - 2), *m);
  CHECK_EQ(c_shape.At(num_c_axes - 1), *n);
}

class FusedMatmulBiasAddReluKernelCache final : public user_op::OpKernelCache {
 public:
  FusedMatmulBiasAddReluKernelCache() {
    // Just for init.
    OF_CUBLAS_CHECK(hipblasLtMatmulDescCreate(&operation_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&cublas_a_desc, HIP_R_32F, 1, 1, 1));
    OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&cublas_b_desc, HIP_R_32F, 1, 1, 1));
    OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&cublas_c_desc, HIP_R_32F, 1, 1, 1));
  }
  ~FusedMatmulBiasAddReluKernelCache() override {
    OF_CUBLAS_CHECK(hipblasLtMatmulDescDestroy(operation_desc));
    OF_CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(cublas_a_desc));
    OF_CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(cublas_b_desc));
    OF_CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(cublas_c_desc));
  }
  hipblasLtMatmulDesc_t operation_desc;
  hipblasLtMatrixLayout_t cublas_a_desc;
  hipblasLtMatrixLayout_t cublas_b_desc;
  hipblasLtMatrixLayout_t cublas_c_desc;
};

std::shared_ptr<FusedMatmulBiasAddReluKernelCache> CreateFusedMatmulBiasAddReluKernelCache() {
  std::shared_ptr<FusedMatmulBiasAddReluKernelCache> cache(new FusedMatmulBiasAddReluKernelCache());
  return cache;
}

void SetCublasMatrixLayout(hipblasLtMatrixLayout_t layout_desc, hipDataType cuda_data_type,
                           hipblasOperation_t cublas_trans, const size_t cublas_m,
                           const size_t cublas_n, int64_t cublas_ld) {
  OF_CUBLAS_CHECK(hipblasLtMatrixLayoutSetAttribute(layout_desc, HIPBLASLT_MATRIX_LAYOUT_TYPE,
                                                   &cuda_data_type, sizeof(cuda_data_type)));
  OF_CUBLAS_CHECK(hipblasLtMatrixLayoutSetAttribute(
      layout_desc, HIPBLASLT_MATRIX_LAYOUT_ROWS, cublas_trans == HIPBLAS_OP_N ? &cublas_m : &cublas_n,
      sizeof(cublas_m)));
  OF_CUBLAS_CHECK(hipblasLtMatrixLayoutSetAttribute(
      layout_desc, HIPBLASLT_MATRIX_LAYOUT_COLS, cublas_trans == HIPBLAS_OP_N ? &cublas_n : &cublas_m,
      sizeof(cublas_m)));
  OF_CUBLAS_CHECK(hipblasLtMatrixLayoutSetAttribute(layout_desc, HIPBLASLT_MATRIX_LAYOUT_LD,
                                                   &cublas_ld, sizeof(cublas_ld)));
}

}  // namespace

template<typename T>
class FusedMatmulBiasAddReluKernel final : public user_op::OpKernel {
 public:
  FusedMatmulBiasAddReluKernel() = default;
  ~FusedMatmulBiasAddReluKernel() override = default;

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  std::shared_ptr<user_op::OpKernelCache> InitOpKernelCache(
      user_op::KernelCacheContext* ctx) const override {
    return CreateFusedMatmulBiasAddReluKernelCache();
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState*,
               const user_op::OpKernelCache* cache) const override {
    const user_op::Tensor* a = ctx->Tensor4ArgNameAndIndex("a", 0);
    const user_op::Tensor* b = ctx->Tensor4ArgNameAndIndex("b", 0);
    const user_op::Tensor* cublas_a = b;
    const user_op::Tensor* cublas_b = a;

    const auto* matmul_cache =
        CHECK_NOTNULL(dynamic_cast<const FusedMatmulBiasAddReluKernelCache*>(cache));

    const user_op::Tensor* bias = ctx->Tensor4ArgNameAndIndex("bias", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    const DataType data_type = out->data_type();

    const auto GetCublasOperation = [](ep::primitive::BlasTransposeType transpose_type) {
      if (transpose_type == ep::primitive::BlasTransposeType::N) {
        return HIPBLAS_OP_N;
      } else if (transpose_type == ep::primitive::BlasTransposeType::T) {
        return HIPBLAS_OP_T;
      } else {
        UNIMPLEMENTED();
        return HIPBLAS_OP_N;
      }
    };

    const auto trans_a = GetBlasTransposeType(ctx, "transpose_a");
    const auto trans_b = GetBlasTransposeType(ctx, "transpose_b");

    size_t m = 0, n = 0, k = 0;
    InferMatmulMNK(a->shape(), b->shape(), out->shape(), trans_a, trans_b, &m, &n, &k);

    /*
    Matmul: A(m, k) x B(k, n) = C(m, n), it follows the row major.
    In cublas, it use column major to compute, Bt(n, k) x At(k, m) = Ct(n, m).
    And Ct matrix follows the column major is equal to C(m, n) which follows the row major.
    */
    const size_t cublas_m = n;
    const size_t cublas_n = m;
    const size_t cublas_k = k;

    const hipblasOperation_t cublas_trans_a = GetCublasOperation(trans_b);
    const hipblasOperation_t cublas_trans_b = GetCublasOperation(trans_a);

    const hipblasComputeType_t cublas_compute_dtype = GetComputeType(data_type);
    const hipDataType cuda_data_type = GetCudaDataType(data_type);

    const double alpha = ctx->Attr<double>("alpha");
    const auto sp_alpha = GetCublasScalarParameter(alpha, cublas_compute_dtype);

    const double beta = 0.0;
    const auto sp_beta = GetCublasScalarParameter(beta, cublas_compute_dtype);

    int64_t cublas_lda = 0;
    if (trans_b == ep::primitive::BlasTransposeType::N) {
      cublas_lda = n;
    } else if (trans_b == ep::primitive::BlasTransposeType::T) {
      cublas_lda = k;
    } else {
      UNIMPLEMENTED();
    }

    int64_t cublas_ldb = 0;
    if (trans_a == ep::primitive::BlasTransposeType::N) {
      cublas_ldb = k;
    } else if (trans_a == ep::primitive::BlasTransposeType::T) {
      cublas_ldb = m;
    } else {
      UNIMPLEMENTED();
    }
    const int64_t cublas_ldc = n;

    OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(
        matmul_cache->operation_desc, CUBLASLT_MATMUL_DESC_COMPUTE_TYPE, &cublas_compute_dtype,
        sizeof(cublas_compute_dtype)));
    // For best performance when using the bias vector, specify beta == 0 and
    // HIPBLASLT_POINTER_MODE_HOST.(from
    // https://docs.nvidia.com/cuda/cublas/index.html#hipblasLtPointerMode_t)
    hipblasLtPointerMode_t mode = HIPBLASLT_POINTER_MODE_HOST;
    OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(
        matmul_cache->operation_desc, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &mode, sizeof(mode)));
    OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmul_cache->operation_desc,
                                                   HIPBLASLT_MATMUL_DESC_TRANSA, &cublas_trans_a,
                                                   sizeof(cublas_trans_a)));
    OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmul_cache->operation_desc,
                                                   HIPBLASLT_MATMUL_DESC_TRANSB, &cublas_trans_b,
                                                   sizeof(cublas_trans_b)));

    // Set as matmul + bias_add + relu.
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_RELU_BIAS;
    OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(
        matmul_cache->operation_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    // Set bias ptr
    const T* bias_ptr = reinterpret_cast<const T*>(bias->dptr());
    OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmul_cache->operation_desc,
                                                   HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias_ptr,
                                                   sizeof(bias_ptr)));

    SetCublasMatrixLayout(matmul_cache->cublas_a_desc, cuda_data_type, cublas_trans_a, cublas_m,
                          cublas_k, cublas_lda);
    SetCublasMatrixLayout(matmul_cache->cublas_b_desc, cuda_data_type, cublas_trans_b, cublas_k,
                          cublas_n, cublas_ldb);
    SetCublasMatrixLayout(matmul_cache->cublas_c_desc, cuda_data_type, HIPBLAS_OP_N, cublas_m,
                          cublas_n, cublas_ldc);

    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();
    OF_CUBLAS_CHECK(hipblasLtMatmul(
        hip_stream->cublas_lt_handle(), matmul_cache->operation_desc, &sp_alpha, cublas_a->dptr(),
        matmul_cache->cublas_a_desc, cublas_b->dptr(), matmul_cache->cublas_b_desc, &sp_beta,
        out->mut_dptr(), matmul_cache->cublas_c_desc, out->mut_dptr(), matmul_cache->cublas_c_desc,
        nullptr, hip_stream->cublas_workspace(), hip_stream->cublas_workspace_size(),
        hip_stream->hip_stream()));
  }
};

#define REGISTER_MATMUL_BIAS_ADD_RELU_KERNEL_GPU(cpp_type, data_type)  \
  REGISTER_USER_KERNEL("fused_matmul_bias_add_relu")                   \
      .SetCreateFn<FusedMatmulBiasAddReluKernel<cpp_type>>()           \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == data_type));

REGISTER_MATMUL_BIAS_ADD_RELU_KERNEL_GPU(double, DataType::kDouble);
REGISTER_MATMUL_BIAS_ADD_RELU_KERNEL_GPU(float, DataType::kFloat);
REGISTER_MATMUL_BIAS_ADD_RELU_KERNEL_GPU(half, DataType::kFloat16);
#if CUDA_VERSION >= 11000
// REGISTER_MATMUL_BIAS_ADD_RELU_KERNEL_GPU(hip_bfloat16, DataType::kBFloat16);
#endif
}  // namespace oneflow
