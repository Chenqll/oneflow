#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/include/primitive/matmul.h"
#include "oneflow/core/common/optional.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include <hip/hip_runtime.h>

namespace oneflow{

namespace{

ep::primitive::BlasTransposeType GetBlasTransposeType(bool transpose) {
    return transpose ? ep::primitive::BlasTransposeType::T : ep::primitive::BlasTransposeType::N;
}

template<typename Context>
ep::primitive::BlasTransposeType GetBlasTransposeType(Context* ctx, const std::string& attr) {
    return GetBlasTransposeType(ctx->template Attr<bool>(attr));
}


Optional<hipDataType> OptCudaDataType(DataType data_type) {
    switch (data_type) {
    case kFloat: return HIP_R_32F;
    case kDouble: return HIP_R_64F;
    case kFloat16: return HIP_R_16F;
#if CUDA_VERSION >= 11000
    case kBFloat16: return HIP_R_16BF;
#endif  // CUDA_VERSION >= 11000
    default: return NullOpt;
    }
}

hipDataType GetCudaDataType(DataType data_type) {
    auto cuda_data_type = OptCudaDataType(data_type);
    CHECK(cuda_data_type.has_value());
    return cuda_data_type.value_or(HIP_R_32F);
}

union CublasScalarParameter {
    double d;
    float s;
};

CublasScalarParameter GetCublasScalarParameter(Scalar scalar, hipDataType compute_type) {
    CublasScalarParameter sp{};
    if (compute_type == HIP_R_64F) {
    sp.d = scalar.Value<double>();
    } else if (compute_type == HIP_R_32F) {
    sp.s = scalar.Value<float>();
    } else {
    UNIMPLEMENTED();
    }
    return sp;
}

hipblasComputeType_t GetComputeType(DataType data_type) {
    switch (data_type) {
    case kFloat: return HIPBLAS_COMPUTE_32F;
    case kDouble: return HIPBLAS_COMPUTE_64F;
    case kFloat16: return HIPBLAS_COMPUTE_16F;
#if CUDA_VERSION >= 11000
    case kBFloat16: return HIPBLAS_COMPUTE_32F_FAST_16BF;
#endif  // CUDA_VERSION >= 11000
    default: UNIMPLEMENTED(); return HIPBLAS_COMPUTE_32F;
    }
}

void InferMatmulMNK(const ShapeView& a_shape, const ShapeView& b_shape, const ShapeView& c_shape,
                    ep::primitive::BlasTransposeType transpose_a, ep::primitive::BlasTransposeType transpose_b, size_t* m, size_t* n, size_t* k) {
    const int64_t num_a_axes = a_shape.NumAxes();
    CHECK_GE(num_a_axes, 2);
    const int64_t num_b_axes = b_shape.NumAxes();
    CHECK_GE(num_b_axes, 2);
    const int64_t num_c_axes = c_shape.NumAxes();
    CHECK_GE(num_c_axes, 2);
    if (transpose_a == ep::primitive::BlasTransposeType::N) {
    *m = a_shape.At(num_a_axes - 2);
    *k = a_shape.At(num_a_axes - 1);
    } else if (transpose_a == ep::primitive::BlasTransposeType::T) {
    *m = a_shape.At(num_a_axes - 1);
    *k = a_shape.At(num_a_axes - 2);
    } else {
    UNIMPLEMENTED();
    }
    if (transpose_b == ep::primitive::BlasTransposeType::N) {
    CHECK_EQ(b_shape.At(num_b_axes - 2), *k);
    *n = b_shape.At(num_b_axes - 1);
    } else if (transpose_b == ep::primitive::BlasTransposeType::T) {
    CHECK_EQ(b_shape.At(num_b_axes - 1), *k);
    *n = b_shape.At(num_b_axes - 2);
    } else {
    UNIMPLEMENTED();
    }
    CHECK_EQ(c_shape.At(num_c_axes - 2), *m);
    CHECK_EQ(c_shape.At(num_c_axes - 1), *n);
}

// class FusedMatmulBiasAddReluKernelState final : public user_op::OpKernelState{
// public: 
//     explicit FusedMatmulBiasAddReluKernelState(user_op::KernelInitContext* ctx){
//         OF_CUBLAS_CHECK(hipblasLtMatmulDescCreate(&operationDesc_, cublas_dtype));
//         OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc_, cublas_dtype, k, m, cublas_lda)); 
//         OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc_, cublas_dtype, n, k, cublas_ldb)); 
//         OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc_, cublas_dtype, n, m, cublas_ldc)); 
//     }
//     todo
//     hipblasLtMatmulDesc_t operationDesc_;
//     hipblasLtMatrixLayout_t Adesc_; 
//     hipblasLtMatrixLayout_t Bdesc_;
//     hipblasLtMatrixLayout_t Cdesc_;
// }; 

} // namespace

template<typename T>
class FusedMatmulBiasAddReluKernel final: public user_op::OpKernel{
public: 
    FusedMatmulBiasAddReluKernel() = default; 
    ~FusedMatmulBiasAddReluKernel() = default; 

    bool AlwaysComputeWhenAllOutputsEmpty() const override {return false; }

private: 
    void Compute(user_op::KernelComputeContext* ctx) const override{
        const user_op::Tensor* a = ctx->Tensor4ArgNameAndIndex("a", 0); 
        const user_op::Tensor* b = ctx->Tensor4ArgNameAndIndex("b", 0); 
        const user_op::Tensor* bias = ctx->Tensor4ArgNameAndIndex("bias", 0); 
        user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0); 
        const DataType data_type = ctx->TensorDesc4ArgNameAndIndex("out", 0)->data_type();
        // TODO: Add check
        const float alpha = 1.0; 
        const float beta = 0.0; 

        const auto GetCublasOperation = [](ep::primitive::BlasTransposeType transpose_type) {
            if (transpose_type == ep::primitive::BlasTransposeType::N) {
              return HIPBLAS_OP_N;
            } else if (transpose_type == ep::primitive::BlasTransposeType::T) {
              return HIPBLAS_OP_T;
            } else {
              UNIMPLEMENTED();
              return HIPBLAS_OP_N;
            }
          };
        
        const auto trans_a = GetBlasTransposeType(ctx, "transpose_a");
        const auto trans_b = GetBlasTransposeType(ctx, "transpose_b");

        size_t m = 0, n = 0, k = 0;
        InferMatmulMNK(a->shape(), b->shape(), out->shape(), trans_a, trans_b, &m, &n, &k);

        const hipblasOperation_t cublas_trans_a = GetCublasOperation(trans_a);
        const hipblasOperation_t cublas_trans_b = GetCublasOperation(trans_b);
        
        const hipblasComputeType_t cublas_compute_dtype = GetComputeType(data_type); 
        const hipDataType cuda_data_type = GetCudaDataType(data_type); 
        
        int cublas_lda = 0;
        if (trans_a == ep::primitive::BlasTransposeType::N) {
            cublas_lda = k;
            printf("Here is None! \n");
        } else if (trans_a == ep::primitive::BlasTransposeType::T) {
            cublas_lda = m;
        } else {
            UNIMPLEMENTED();
        }
        
        int cublas_ldb = 0;
        if (trans_b == ep::primitive::BlasTransposeType::N) {
            cublas_ldb = n;
            printf("Here is None! \n");
        } else if (trans_b == ep::primitive::BlasTransposeType::T) {
            cublas_ldb = k;
        } else {
            UNIMPLEMENTED();
        }

        const int cublas_ldc = n;
        #if CUDA_VERSION >= 11000
        hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;
        #else
        hipblasGemmAlgo_t algo =
            (data_type == DataType::kFloat16) ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT;
        #endif

        hipblasLtMatmulDesc_t operationDesc = NULL;
        OF_CUBLAS_CHECK(hipblasLtMatmulDescCreate(&operationDesc, cublas_compute_dtype, cuda_data_type));
        // OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &cublas_trans_a, sizeof(cublas_trans_a)));
        // OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &cublas_trans_b, sizeof(cublas_trans_b)));
        hipblasOperation_t trans = HIPBLAS_OP_N;
        OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(trans)));
        OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &trans, sizeof(trans)));
        
        // Set as matmul + bias_add + relu. 
        hipblasLtEpilogue_t epilogue;
        // epilogue = HIPBLASLT_EPILOGUE_RELU_BIAS;
        epilogue = HIPBLASLT_EPILOGUE_RELU;
        OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue))); 
        
        // T* relu_mask_ptr = reinterpret_cast<T*>(relu_mask->mut_dptr()); 
        // long reluMaskLd = n;
        // // Set relu mask ptr in cublas aux pointer. 
        // OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
        //     &relu_mask_ptr, sizeof(relu_mask_ptr)));
        
        // // Set relu mask leading dimension. 
        // OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
        //                                &reluMaskLd, sizeof(reluMaskLd));
        
        // Set bias ptr
        // const T* bias_ptr = reinterpret_cast<const T*>(bias->dptr()); 
        // OF_CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(
        //     operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias_ptr, sizeof(bias_ptr)));
        
        hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, cuda_data_type, m, k, cublas_lda)); 
        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, cuda_data_type, k, n, cublas_ldb)); 
        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, cuda_data_type, m, n, cublas_ldc));
        // todo! 

        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, cuda_data_type, k, m, cublas_lda)); 
        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, cuda_data_type, n, k, cublas_ldb)); 
        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, cuda_data_type, n, m, cublas_ldc)); 

        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, cuda_data_type, m, k, k)); 
        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, cuda_data_type, k, n, n)); 
        // OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, cuda_data_type, m, n, n)); 
        
        OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, cuda_data_type, n, k, n));
        OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, cuda_data_type, k, m, k)); 
        OF_CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, cuda_data_type, n, m, n)); 

        printf("M is %d \n", m);
        printf("N is %d \n", n);
        printf("K is %d \n", k);
        printf("LDA is %d \n", cublas_lda);
        printf("LDB is %d \n", cublas_ldb);
        printf("LDC is %d \n", cublas_ldc);



        // OF_CUBLAS_CHECK(hipblasLtMatmul(ctx->stream()->As<ep::CudaStream>()->cublas_lt_handle(),
        //                                operationDesc,
        //                                &alpha,
        //                                reinterpret_cast<const T*>(a->dptr()),
        //                                 Adesc,
        //                                 reinterpret_cast<const T*>(b->dptr()),
        //                                 Bdesc,
        //                                 &beta,
        //                                 reinterpret_cast<T*>(out->mut_dptr()),
        //                                 Cdesc,
        //                                 reinterpret_cast<T*>(out->mut_dptr()),
        //                                 Cdesc,
        //                                 NULL,
        //                                 NULL,
        //                                 0,
        //                                 0));

        OF_CUBLAS_CHECK(hipblasLtMatmul(ctx->stream()->As<ep::CudaStream>()->cublas_lt_handle(),
                                       operationDesc,
                                       &alpha,
                                        reinterpret_cast<const T*>(b->dptr()),
                                        Bdesc,
                                        reinterpret_cast<const T*>(a->dptr()),
                                        Adesc,
                                        &beta,
                                        reinterpret_cast<T*>(out->mut_dptr()),
                                        Cdesc,
                                        reinterpret_cast<T*>(out->mut_dptr()),
                                        Cdesc,
                                        NULL,
                                        NULL,
                                        0,
                                        0));
        // TODO: whether to destroy? 
        // 放到kernel state. 
        OF_CUBLAS_CHECK(hipblasLtMatmulDescDestroy(operationDesc));
        OF_CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(Adesc)); 
        OF_CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(Bdesc)); 
        OF_CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(Cdesc)); 
    }
}; 

#define REGISTER_MATMUL_BIAS_ADD_RELU_KERNEL_GPU(cpp_type, data_type)                                     \
  REGISTER_USER_KERNEL("fused_matmul_bias_add_relu").SetCreateFn<FusedMatmulBiasAddReluKernel<cpp_type>>().SetIsMatchedHob( \
    (user_op::HobDeviceType() == DeviceType::kCUDA)                                        \
    && (user_op::HobDataType("out", 0) == data_type));

REGISTER_MATMUL_BIAS_ADD_RELU_KERNEL_GPU(float, DataType::kFloat); 

} // namespace oneflow  